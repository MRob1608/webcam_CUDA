#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "conversion_CUDA.cuh"
#include "globals.h"

// Kernel to convert an image from YUYV format to BGRA
__global__ void yuyv_to_bgr_CUDA(unsigned char *yuyv, unsigned char *rgb, int height, int width)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= height * width / 2) {
        return;
    }

    int y, v, u;
    float r, g, b;

    y = yuyv[4*tid + 0]; //y0
    u = yuyv[4*tid + 1]; //u0
    v = yuyv[4*tid + 3]; //v0

    r = y + 1.4065 * (v - 128);			     //r0
    g = y - 0.3455 * (u - 128) - 0.7169 * (v - 128); //g0
    b = y + 1.1790 * (u - 128);			     //b0

    if (r < 0)
	r = 0;
    else if (r > 255)
	r = 255;
    if (g < 0)
	g = 0;
    else if (g > 255)
	g = 255;
    if (b < 0)
	b = 0;
    else if (b > 255)
	b = 255;

    rgb[8*tid + 0] = (unsigned char)b;
    rgb[8*tid + 1] = (unsigned char)g;
    rgb[8*tid + 2] = (unsigned char)r;
    rgb[8*tid + 3] = (unsigned char)0;


    //second pixel
    u = yuyv[4*tid + 1]; //u0
    y = yuyv[4*tid + 2]; //y1
    v = yuyv[4*tid + 3]; //v0

    r = y + 1.4065 * (v - 128);			     //r1
    g = y - 0.3455 * (u - 128) - 0.7169 * (v - 128); //g1
    b = y + 1.1790 * (u - 128);			     //b1

    if (r < 0)
	r = 0;
    else if (r > 255)
	r = 255;
    if (g < 0)
	g = 0;
    else if (g > 255)
	g = 255;
    if (b < 0)
	b = 0;
    else if (b > 255)
	b = 255;

    rgb[8*tid + 4] = (unsigned char)b;
    rgb[8*tid + 5] = (unsigned char)g;
    rgb[8*tid + 6] = (unsigned char)r;
    rgb[8*tid + 7] = (unsigned char)0;
}

//Kernel to mirror the image
__global__ void mirror_image_kernel(unsigned char* rgb, unsigned char* mirrored_rgb, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int src_idx = ((width * y) + x) * 4;
    int dest_idx = ((width * y) + width - x - 1) * 4;   //position of the new pixel


    mirrored_rgb[dest_idx] = rgb[src_idx];
    mirrored_rgb[dest_idx+1] = rgb[src_idx+1];
    mirrored_rgb[dest_idx+2] = rgb[src_idx+2];
    mirrored_rgb[dest_idx+3] = rgb[src_idx+3];
}

//Wrapper for the mirroring kernel
void mirror_image_gpu(unsigned char* rgb, int width, int height) {
    dim3 blockSize(16, 16);  
    dim3 gridSize(
      (width + blockSize.x - 1) / blockSize.x,   //ceil of width / blocksize
      (height + blockSize.y - 1) / blockSize.y   //ceil of height / blocksize
    );

    hipMemcpy(device_rgb, rgb, width * height * 4, hipMemcpyHostToDevice);

    mirror_image_kernel<<<gridSize, blockSize>>>(device_rgb, device_mirrored_rgb, width, height);

    hipMemcpy(rgb, device_mirrored_rgb, width * height * 4, hipMemcpyDeviceToHost);
}
