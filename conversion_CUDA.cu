#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "conversion_CUDA.cuh"



__global__ void yuyv_to_bgr_CUDA(unsigned char *yuyv, unsigned char *rgb, int height, int width)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= height * width / 2) {
        return;
    }

    int y, v, u;
    float r, g, b;

    y = yuyv[4*tid + 0]; //y0
    u = yuyv[4*tid + 1]; //u0
    v = yuyv[4*tid + 3]; //v0

    r = y + 1.4065 * (v - 128);			     //r0
    g = y - 0.3455 * (u - 128) - 0.7169 * (v - 128); //g0
    b = y + 1.1790 * (u - 128);			     //b0

    if (r < 0)
	r = 0;
    else if (r > 255)
	r = 255;
    if (g < 0)
	g = 0;
    else if (g > 255)
	g = 255;
    if (b < 0)
	b = 0;
    else if (b > 255)
	b = 255;

    rgb[8*tid + 0] = (unsigned char)b;
    rgb[8*tid + 1] = (unsigned char)g;
    rgb[8*tid + 2] = (unsigned char)r;
    rgb[8*tid + 3] = (unsigned char)0;


    //second pixel
    u = yuyv[4*tid + 1]; //u0
    y = yuyv[4*tid + 2]; //y1
    v = yuyv[4*tid + 3]; //v0

    r = y + 1.4065 * (v - 128);			     //r1
    g = y - 0.3455 * (u - 128) - 0.7169 * (v - 128); //g1
    b = y + 1.1790 * (u - 128);			     //b1

    if (r < 0)
	r = 0;
    else if (r > 255)
	r = 255;
    if (g < 0)
	g = 0;
    else if (g > 255)
	g = 255;
    if (b < 0)
	b = 0;
    else if (b > 255)
	b = 255;

    rgb[8*tid + 4] = (unsigned char)b;
    rgb[8*tid + 5] = (unsigned char)g;
    rgb[8*tid + 6] = (unsigned char)r;
    rgb[8*tid + 7] = (unsigned char)0;
}

/*

__global__ void swap(unsigned char* a, unsigned char * b) {
    unsigned char tmp = *b;
    *b = *a;
    *a = tmp;
}

__global__ void mirror_image(unsigned char* rgb, int height, int width) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= width * height /2) return;
    int bytes_per_line = width *4;
    int offset = tid % width;
    int num_line = (int) (tid / width);
    for ( int i = 0; i < height; i++) {
        int idx = i * bytes_per_line;
        for (int j = 0; j < width*2; j+= 4) {
            swap((unsigned char *)&rgb[idx+j], (unsigned char *)&rgb[idx+bytes_per_line-4-j]);
            swap((unsigned char *)&rgb[idx+j+1], (unsigned char *)&rgb[idx+bytes_per_line-3-j]);
            swap((unsigned char *)&rgb[idx+j+2], (unsigned char *)&rgb[idx+bytes_per_line-2-j]);
            swap((unsigned char *)&rgb[idx+j+3], (unsigned char *)&rgb[idx+bytes_per_line-1-j]);
        }
    }
}

*/