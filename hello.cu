#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_cuda() {
    printf("Hello from GPU!\n");
}

int main() {
    hello_cuda<<<1, 1>>>();
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }

    hipDeviceSynchronize();  // forza completamento kernel
    fflush(stdout);           // forza stampa terminale
    return 0;
}
