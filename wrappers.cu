#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "capture_camera.h"
#include "image_manipulation.cuh"
#include "conversion_CUDA.cuh"
#include "conversion.h"
#include "globals.h"




void convert_yuyv_to_bgra(camera_t* camera, char* rgb, int use_gpu) { 
  if (use_gpu) {


        int num_pixels = camera->width * camera->height;
        int num_threads = 256;
        int num_blocks = (num_pixels / 2 + num_threads - 1) / num_threads;

        hipMemcpy(device_yuyv, camera->head.start, camera->width * camera->height * 2, hipMemcpyHostToDevice);

        yuyv_to_bgr_CUDA<<<num_blocks, num_threads>>>(device_yuyv, device_rgb, camera->height, camera->width);

        hipMemcpy(rgb, device_rgb, camera->width * camera->height * 4, hipMemcpyDeviceToHost);

      } else {
        yuyv_to_bgr(camera->head.start,(unsigned char *)rgb,camera->width, camera->height);
      }
}


void apply_edge_detection(char* rgb, int width, int height) {
  hipMemcpy(device_rgb, (unsigned char *)rgb, width * height * 4, hipMemcpyHostToDevice);

  int num_pixels = width * height;
  int num_threads = 256;
  int num_blocks = (num_pixels + num_threads - 1) / num_threads;
        
  gray_scale_conversion<<<num_blocks, num_threads>>>(device_rgb, device_gray, width, height);

  dim3 blockSize(16, 16);  // 256 thread per blocco
  dim3 gridSize(
      (width + blockSize.x - 1) / blockSize.x,   // ceil(width / 16)
      (height + blockSize.y - 1) / blockSize.y   // ceil(height / 16)
  );


  edge_detection_overlay<<<gridSize, blockSize>>>(device_rgb, device_output, device_gray, width, height);

  hipMemcpy((unsigned char*)rgb, device_output, width * height * 4, hipMemcpyDeviceToHost);

}




void apply_optical_flow(char* rgb, char* prev_rgb, int width, int height) {

  dim3 block(16, 16);
  dim3 grid((width + 15) / 16, (height + 15) / 16);

  int num_pixels = width * height;
  int num_threads = 256;
  int num_blocks = (num_pixels + num_threads - 1) / num_threads;

  hipMemcpy(device_rgb, (unsigned char *)rgb, width * height * 4, hipMemcpyHostToDevice);
  gray_scale_conversion<<<num_blocks, num_threads>>>(device_rgb, device_gray, width, height);

  gaussian_blur3<<<grid, block>>>(device_gray, device_blur, width, height);

  hipMemcpy(device_prev_rgb, (unsigned char *)prev_rgb, width * height * 4, hipMemcpyHostToDevice);
  gray_scale_conversion<<<num_blocks, num_threads>>>(device_prev_rgb, device_prev_gray, width, height);

  gaussian_blur3<<<grid, block>>>(device_prev_gray, device_prev_blur, width, height);

  compute_derivatives<<<grid, block>>>(device_prev_blur, device_blur ,d_Ix, d_Iy, d_It, width, height);

  size_t fsize = width * height * sizeof(float);

  int num_iterations = 100;
  float alpha = 10.0f;

  for (int i = 0; i < num_iterations; i++) {
      // 1. Calcola media locale
      average_uv<<<grid, block>>>(
          d_u, d_v,
          d_u_avg, d_v_avg,
          width, height
      );

            // 2. Aggiorna u, v
            update_uv<<<grid, block>>>(
                d_Ix, d_Iy, d_It,
                d_u_avg, d_v_avg,
                d_u, d_v,
                alpha,
                width, height
            );
        }

        compute_flow_magnitude<<<grid, block>>>(d_u, d_v, mag, width, height);

        float* h_mag = (float*)malloc(fsize);
        hipMemcpy(h_mag, mag, fsize, hipMemcpyDeviceToHost);

        float* h_u = (float*)malloc(fsize);
        float* h_v = (float*)malloc(fsize);
        hipMemcpy(h_u, d_u, fsize, hipMemcpyDeviceToHost);
        hipMemcpy(h_v, d_v, fsize, hipMemcpyDeviceToHost);

        float best_magnitude = 0.0f;
        int best_x = 0, best_y = 0;
        int block_size = 32;

        for (int by = 0; by < height; by += block_size) {
            for (int bx = 0; bx < width; bx += block_size) {
                float sum_u = 0.0f;
                float sum_v = 0.0f;
                int count = 0;

                for (int y = 0; y < block_size; y++) {
                    for (int x = 0; x < block_size; x++) {
                        int px = bx + x;
                        int py = by + y;
                        if (px >= width || py >= height) continue;
                        int idx = py * width + px;
                        sum_u += h_u[idx];
                        sum_v += h_v[idx];
                        count++;
                    }
                }

                if (count > 0) {
                    float avg_u = sum_u / count;
                    float avg_v = sum_v / count;
                    float magnitude = sqrtf(avg_u * avg_u + avg_v * avg_v);
                    if (magnitude > best_magnitude) {
                        best_magnitude = magnitude;
                        best_x = bx + block_size / 2;
                        best_y = by + block_size / 2;
                    }
                }
            }
        }

        free(h_u);
        free(h_v);
        free(h_mag);

        if (best_magnitude < 20.0f) {
            best_x = prev_x;
            best_y = prev_y;
        }

        int box_size = 16 + fminf(best_magnitude * 2, 200.0f); 

        float square_alpha = 0.2f;

        float filtered_x = square_alpha * best_x + (1.0f - square_alpha) * prev_x;
        float filtered_y = square_alpha * best_y + (1.0f - square_alpha) * prev_y;

        prev_x = filtered_x;
        prev_y = filtered_y;

        draw_square((unsigned char*)rgb, width, height, (int)filtered_x, (int)filtered_y, box_size);
        
}

void scale_image_cn(unsigned char* rgb, int base_width, int base_height,unsigned char* scaled_image ,int window_width, int window_height) {

    hipMemcpy(device_rgb, rgb, base_width * base_height * 4, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((window_width + 15) / 16, (window_height + 15) / 16);

    scale_image_cn_kernel<<<gridSize, blockSize>>>(device_rgb, base_width, base_height ,device_scaled_rgb, window_width, window_height);

    hipMemcpy(scaled_image, device_scaled_rgb, window_height * window_width * 4, hipMemcpyDeviceToHost);

}

void scale_image_bilinear(unsigned char* rgb, int base_width, int base_height,unsigned char* scaled_image ,int window_width, int window_height) {
    hipMemcpy(device_rgb, rgb, base_width * base_height * 4, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((window_width + 15) / 16, (window_height + 15) / 16);

    scale_image_bilinear_kernel<<<gridSize, blockSize>>>(device_rgb, device_scaled_rgb, base_width, base_height, window_width, window_height);

    image_sharpen<<<gridSize, blockSize>>>(device_scaled_rgb, device_sharpened_rgb, window_width, window_height);

    hipMemcpy(scaled_image, device_sharpened_rgb, window_height * window_width * 4, hipMemcpyDeviceToHost);
}


void alloc_conversion(camera_t* camera) {
    hipMalloc(&device_yuyv, camera->width * camera->height * 2);
    hipMalloc(&device_rgb, camera->width * camera->height * 4);
}

void free_conversion(void) {
    hipFree(device_rgb);
    hipFree(device_yuyv);
}

void alloc_Edge(camera_t* camera) {
    if (!GPU) {
      hipMalloc(&device_rgb, camera->width * camera->height * 4);
    }
    hipMalloc(&device_gray, camera->width * camera->height);
    hipMalloc(&device_output, camera->width * camera->height * 4);
}

void free_Edge(void) {
    hipFree(device_rgb);
    hipFree(device_gray);
    hipFree(device_output);
}

void alloc_Optical(camera_t* camera) {
    size_t derivative_size = camera->width * camera->height * sizeof(float);
    size_t fsize = camera->width * camera->height * sizeof(float);

    if (!GPU) {
      hipMalloc(&device_rgb, camera->width * camera->height * 4);
    }
    if (!EDGE_DET) {
      hipMalloc(&device_gray, camera->width * camera->height);
    }

    hipMalloc(&device_prev_gray, camera->width * camera->height);
    hipMalloc(&device_prev_rgb, camera->width * camera->height *4);
    hipMalloc(&d_Ix, derivative_size);
    hipMalloc(&d_Iy, derivative_size);
    hipMalloc(&d_It, derivative_size);
    hipMalloc(&d_u, fsize);
    hipMalloc(&d_v, fsize);
    hipMemset(d_u, 0, fsize);  
    hipMemset(d_v, 0, fsize);
    hipMalloc(&d_u_avg, fsize);
    hipMalloc(&d_v_avg, fsize);
    hipMalloc(&mag,camera->width * camera->height * sizeof(float));
    hipMalloc(&device_blur, camera->width * camera->height);
    hipMalloc(&device_prev_blur, camera->width * camera->height);
}


void free_Optical(void) {
    hipFree(device_rgb);
    hipFree(device_prev_rgb);
    hipFree(device_gray);
    hipFree(device_prev_gray);
    hipFree(d_It);
    hipFree(d_Ix);
    hipFree(d_Iy);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_u_avg);
    hipFree(d_v_avg);
    hipFree(mag);
    hipFree(device_blur);
    hipFree(device_prev_blur);
}