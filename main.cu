#include "hip/hip_runtime.h"
extern "C" {
#include "capture_camera.h"
#include "interface.h"
#include "conversion.h"
}
#include "conversion_CUDA.cuh"
#include "image_manipulation.cuh"

#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <errno.h>

#include <fcntl.h>
#include <sys/ioctl.h>
#include <sys/mman.h>
#include <asm/types.h>
#include <linux/videodev2.h>

#include <sys/time.h>
#include <sys/types.h>
#include <unistd.h>

#include <time.h>
#include <X11/Xlib.h>
#include <X11/Xatom.h>

extern Display* display;
extern Window window;
extern GC gc;
XEvent event;

int GPU = 1;
int EDGE_DET = 0;
int OPTICAL = 1;

unsigned char* device_rgb, *device_gray, *device_output, *device_yuyv, *device_blur;

unsigned char* device_prev_rgb, *device_prev_gray, *device_prev_blur;

float* d_Ix, *d_It, *d_Iy, *d_u, *d_v, *d_u_avg, *d_v_avg, *mag;


void draw_square(
    unsigned char* image,
    int width, int height,
    int center_x, int center_y,
    int size
) {
    int half = size / 2;
    for (int y = -half; y <= half; y++) {
        for (int x = -half; x <= half; x++) {
            int px = center_x + x;
            int py = center_y + y;
            if (px >= 0 && px < width && py >= 0 && py < height && ((x == -half or x ==half) || (y == -half or y ==half))) {
                int idx = (py * width + px) * 4; // BGRA
                image[idx + 0] = 0;   // B
                image[idx + 1] = 255; // G
                image[idx + 2] = 0;   // R
                image[idx + 3] = 0; // A
            }
        }
    }
}

void convert_yuyv_to_bgra(camera_t* camera, char* rgb, int use_gpu) { 
  if (use_gpu) {


        int num_pixels = camera->width * camera->height;
        int num_threads = 256;
        int num_blocks = (num_pixels / 2 + num_threads - 1) / num_threads;

        hipMemcpy(device_yuyv, camera->head.start, camera->width * camera->height * 2, hipMemcpyHostToDevice);

        yuyv_to_bgr_CUDA<<<num_blocks, num_threads>>>(device_yuyv, device_rgb, camera->height, camera->width);

        hipMemcpy(rgb, device_rgb, camera->width * camera->height * 4, hipMemcpyDeviceToHost);

      } else {
        yuyv_to_bgr(camera->head.start,(unsigned char *)rgb,camera->width, camera->height);
      }
}

void apply_edge_detection(char* rgb, int width, int height) {
  hipMemcpy(device_rgb, (unsigned char *)rgb, width * height * 4, hipMemcpyHostToDevice);

  int num_pixels = width * height;
  int num_threads = 256;
  int num_blocks = (num_pixels + num_threads - 1) / num_threads;
        
  gray_scale_conversion<<<num_blocks, num_threads>>>(device_rgb, device_gray, width, height);

  dim3 blockSize(16, 16);  // 256 thread per blocco
  dim3 gridSize(
      (width + blockSize.x - 1) / blockSize.x,   // ceil(width / 16)
      (height + blockSize.y - 1) / blockSize.y   // ceil(height / 16)
  );


  edge_detection_overlay<<<gridSize, blockSize>>>(device_rgb, device_output, device_gray, width, height);

  hipMemcpy((unsigned char*)rgb, device_output, width * height * 4, hipMemcpyDeviceToHost);

}


void apply_optical_flow(char* rgb, char* prev_rgb, int width, int height) {

  dim3 block(16, 16);
  dim3 grid((width + 15) / 16, (height + 15) / 16);

  int num_pixels = width * height;
  int num_threads = 256;
  int num_blocks = (num_pixels + num_threads - 1) / num_threads;

  hipMemcpy(device_rgb, (unsigned char *)rgb, width * height * 4, hipMemcpyHostToDevice);
  gray_scale_conversion<<<num_blocks, num_threads>>>(device_rgb, device_gray, width, height);

  //blur_image<<<grid, block>>>(device_gray, device_blur, width, height);

  hipMemcpy(device_prev_rgb, (unsigned char *)prev_rgb, width * height * 4, hipMemcpyHostToDevice);
  gray_scale_conversion<<<num_blocks, num_threads>>>(device_prev_rgb, device_prev_gray, width, height);

  //blur_image<<<grid, block>>>(device_prev_gray, device_prev_blur, width, height);

  compute_derivatives<<<grid, block>>>(device_prev_gray, device_gray ,d_Ix, d_Iy, d_It,width, height);

  size_t fsize = width * height * sizeof(float);

  int num_iterations = 100;
  float alpha = 15.0f;

  for (int i = 0; i < num_iterations; i++) {
      // 1. Calcola media locale
      average_uv<<<grid, block>>>(
          d_u, d_v,
          d_u_avg, d_v_avg,
          width, height
      );

            // 2. Aggiorna u, v
            update_uv<<<grid, block>>>(
                d_Ix, d_Iy, d_It,
                d_u_avg, d_v_avg,
                d_u, d_v,
                alpha,
                width, height
            );
        }

        compute_flow_magnitude<<<grid, block>>>(d_u, d_v, mag, width, height);

        float* h_mag = (float*)malloc(fsize);
        hipMemcpy(h_mag, mag, fsize, hipMemcpyDeviceToHost);

        float* h_u = (float*)malloc(fsize);
        float* h_v = (float*)malloc(fsize);
        hipMemcpy(h_u, d_u, fsize, hipMemcpyDeviceToHost);
        hipMemcpy(h_v, d_v, fsize, hipMemcpyDeviceToHost);

        float best_magnitude = 0.0f;
        int best_x = 0, best_y = 0;
        int block_size = 32;

        for (int by = 0; by < height; by += block_size) {
            for (int bx = 0; bx < width; bx += block_size) {
                float sum_u = 0.0f;
                float sum_v = 0.0f;
                int count = 0;

                for (int y = 0; y < block_size; y++) {
                    for (int x = 0; x < block_size; x++) {
                        int px = bx + x;
                        int py = by + y;
                        if (px >= width || py >= height) continue;
                        int idx = py * width + px;
                        sum_u += h_u[idx];
                        sum_v += h_v[idx];
                        count++;
                    }
                }

                if (count > 0) {
                    float avg_u = sum_u / count;
                    float avg_v = sum_v / count;
                    float magnitude = sqrtf(avg_u * avg_u + avg_v * avg_v);
                    if (magnitude > best_magnitude) {
                        best_magnitude = magnitude;
                        best_x = bx + block_size / 2;
                        best_y = by + block_size / 2;
                    }
                }
            }
        }

        free(h_u);
        free(h_v);

        int box_size = 16 + best_magnitude * 2; // minimo 16, massimo 48 se avg ≈ 1.0

        draw_square((unsigned char*)rgb, width, height, best_x, best_y, box_size);
        
}




int main(int argc, char** argv)
{
  if (argc != 2) {
    printf("usage: <executable> <camera_device_name> - eg ./camera_capture /dev/video0\n");
    return -1;
  }

  printf("opening camera device [ %s ]\n", argv[1]);
  camera_t* camera = camera_open("/dev/video0", 640, 480);  //640 480

  init_x11(640,480);
  Atom wm_delete_window = XInternAtom(display, "WM_DELETE_WINDOW", False);

  camera_init(camera);
  camera_start(camera);


  struct timespec start_time, end_time;
  clock_gettime(CLOCK_MONOTONIC, &start_time);

  struct timeval timeout;
  timeout.tv_sec = 0;
  timeout.tv_usec = 100000;
  char image_name[1024];
  char* prev_rgb =  (char*)malloc(camera->width * camera->height * 4);

  if(GPU) {
    hipMalloc(&device_yuyv, camera->width * camera->height * 2);
    hipMalloc(&device_rgb, camera->width * camera->height * 4);
  }

  if (EDGE_DET) {
    if (!GPU) {
      hipMalloc(&device_rgb, camera->width * camera->height * 4);
    }
    hipMalloc(&device_gray, camera->width * camera->height);
    hipMalloc(&device_output, camera->width * camera->height * 4);
  }

  if (OPTICAL) {
    size_t derivative_size = camera->width * camera->height * sizeof(float);
    size_t fsize = camera->width * camera->height * sizeof(float);

    if (!GPU) {
      hipMalloc(&device_rgb, camera->width * camera->height * 4);
    }
    if (!EDGE_DET) {
      hipMalloc(&device_gray, camera->width * camera->height);
    }

    hipMalloc(&device_prev_gray, camera->width * camera->height);
    hipMalloc(&device_prev_rgb, camera->width * camera->height);
    hipMalloc(&d_Ix, derivative_size);
    hipMalloc(&d_Iy, derivative_size);
    hipMalloc(&d_It, derivative_size);
    hipMalloc(&d_u, fsize);
    hipMalloc(&d_v, fsize);
    hipMemset(d_u, 0, fsize);  
    hipMemset(d_v, 0, fsize);
    hipMalloc(&d_u_avg, fsize);
    hipMalloc(&d_v_avg, fsize);
    hipMalloc(&mag,camera->width * camera->height * sizeof(float));
  }

  hipMalloc(&device_blur, camera->width * camera->height);
  hipMalloc(&device_prev_blur, camera->width * camera->height);

  int i = 0;

  printf("cartes"); fflush(stdout);

  while(1) {
    while (XPending(display)) {
        XNextEvent(display, &event);
        if (event.type == ClientMessage) {
            if ((Atom)event.xclient.data.l[0] == wm_delete_window) {
                printf("\nFinestra chiusa\n");
                goto exit_loop;
            }
        }
    }


    if (camera_frame(camera, timeout)>0) {
      sprintf(image_name, "image-%05d.pgm", i);
      printf("\racquiring frame [ %05d ]", i);
      fflush(stdout);
      char* rgb =  (char*)malloc(camera->width * camera->height * 4);
      
      convert_yuyv_to_bgra(camera,rgb, GPU);


      if (EDGE_DET) {
        apply_edge_detection(rgb, camera->width, camera->height);
      }

      if (OPTICAL && prev_rgb != NULL) {
        apply_optical_flow(rgb, prev_rgb, camera->width, camera->height);
      }

      
      mirror_image((unsigned char*)rgb, camera->height, camera->width);
      //savePGM(camera, image_name);
      display_frame((unsigned char*)rgb,camera->width, camera->height);
      memcpy(prev_rgb, rgb, camera->width * camera->height * 4);
      free(rgb);
    }
    i++;
  }
  exit_loop:
  XDestroyWindow(display, window);
  XCloseDisplay(display);

  clock_gettime(CLOCK_MONOTONIC, &end_time); // dopo il ciclo
  double elapsed_sec = end_time.tv_sec - start_time.tv_sec +
                     (end_time.tv_nsec - start_time.tv_nsec) / 1e9;

  printf("Captured %d frames in %.2f seconds (%.2f FPS)\n", i, elapsed_sec, i / elapsed_sec);
  printf("done!\n");
  camera_frame(camera, timeout);

  if (GPU) {
    hipFree(device_rgb);
    hipFree(device_yuyv);
  }
  
  if(EDGE_DET) {
    hipFree(device_rgb);
    hipFree(device_gray);
    hipFree(device_output);
  }

  if (OPTICAL) {
    hipFree(device_rgb);
    hipFree(device_prev_rgb);
    hipFree(device_gray);
    hipFree(device_prev_gray);
    hipFree(d_It);
    hipFree(d_Ix);
    hipFree(d_Iy);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_u_avg);
    hipFree(d_v_avg);
    hipFree(mag);
  } 

  hipFree(device_blur);
  hipFree(device_prev_blur);
  


  printf("closing\n");
  camera_stop(camera);
  camera_finish(camera);
  camera_close(camera);
  return 0;
}

